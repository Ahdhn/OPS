#include "hip/hip_runtime.h"
/*
* Open source copyright declaration based on BSD open source template:
* http://www.opensource.org/licenses/bsd-license.php
*
* This file is part of the OPS distribution.
*
* Copyright (c) 2013, Mike Giles and others. Please see the AUTHORS file in
* the main source directory for a full list of copyright holders.
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
* * Redistributions of source code must retain the above copyright
* notice, this list of conditions and the following disclaimer.
* * Redistributions in binary form must reproduce the above copyright
* notice, this list of conditions and the following disclaimer in the
* documentation and/or other materials provided with the distribution.
* * The name of Mike Giles may not be used to endorse or promote products
* derived from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY Mike Giles ''AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
* WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
* DISCLAIMED. IN NO EVENT SHALL Mike Giles BE LIABLE FOR ANY
* DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
* (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
* LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
* ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
* SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

/** @brief ops mpi+cuda run-time support routines
  * @author Gihan Mudalige, Istvan Reguly
  * @details Implements the runtime support routines for the OPS mpi+cuda
 * backend
  */

#include <ops_cuda_rt_support.h>
#include <ops_mpi_core.h>

#ifdef __cplusplus
extern "C" {
#endif

int halo_buffer_size = 0;
char *halo_buffer_d = NULL;

__global__ void ops_cuda_packer_1(const char *__restrict src,
                                  char *__restrict dest, int count, int len,
                                  int stride, int hybrid_offset) {
  int idx = hybrid_offset + blockIdx.x * blockDim.x + threadIdx.x;
  int block = idx / len;
  if (idx < count * len) {
    dest[idx] = src[stride * block + idx % len];
  }
}

__global__ void ops_cuda_packer_1_soa(const char *__restrict src,
                                  char *__restrict dest, int count, int len,
                                  int stride, int hybrid_offset, int dim, int size) {
  int idx = hybrid_offset + blockIdx.x * blockDim.x + threadIdx.x;
  int block = idx / len;
  if (idx < count * len) {
    for (int d=0; d<dim; d++) {   
      dest[idx*dim+d] = src[stride * block + idx % len + d * size];
    }
  }
}

__global__ void ops_cuda_unpacker_1(const char *__restrict src,
                                    char *__restrict dest, int count, int len,
                                    int stride) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int block = idx / len;
  if (idx < count * len) {
    dest[stride * block + idx % len] = src[idx];
  }
}

__global__ void ops_cuda_unpacker_1_soa(const char *__restrict src,
                                    char *__restrict dest, int count, int len,
                                    int stride, int dim, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int block = idx / len;
  if (idx < count * len) {
    for (int d=0; d<dim; d++) {   
      dest[stride * block + idx % len + d * size] = src[idx*dim + d];
    }
  }
}


__global__ void ops_cuda_packer_4(const int *__restrict src,
                                  int *__restrict dest, int count, int len,
                                  int stride, int hybrid_offset) {
  int idx = hybrid_offset + blockIdx.x * blockDim.x + threadIdx.x;
  int block = idx / len;
  if (idx < count * len) {
    dest[idx] = src[stride * block + idx % len];
  }
}

__global__ void ops_cuda_unpacker_4(const int *__restrict src,
                                    int *__restrict dest, int count, int len,
                                    int stride) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int block = idx / len;
  if (idx < count * len) {
    dest[stride * block + idx % len] = src[idx];
  }
}

extern "C" int ops_hybrid_get_clean_cpu(ops_dat dat);
extern "C" void ops_pack_hybrid_cpu(ops_dat dat, const int src_offset, char *__restrict dest,
               int halo_blocklength, int halo_stride, int new_count);
extern "C" void ops_unpack_hybrid_cpu(ops_dat dat, const int dest_offset, const char *__restrict src,
                 int halo_blocklength, int halo_stride, int new_count);

void ops_pack(ops_dat dat, const int src_offset, char *__restrict dest,
              const ops_int_halo *__restrict halo) {

  if (dat->dirty_hd == 1) {
    ops_upload_dat(dat);
    dat->dirty_hd = 0;
  }
  //printf("Halo exchange of %s: offset %d, count %d, blocklen %d stride %d\n",dat->name, src_offset, halo->count, halo->blocklength, halo->stride);

  int hybrid_offset = 0; //if hybrid, where is the split betwen cpu and gpu. -1 if last dim 

  //With hybrid, we need to split the halo gather based on who has what
  //CPU has up to dirty_from_h. With overlapping clean regions, should we prefer CPU or GPU??
  if (ops_hybrid) {
    if (OPS_gpu_direct) {ops_printf("Error: hybrid execution and GPU Direct combination unsupported\n"); exit(-1);}
    int cpu_clean = ops_hybrid_get_clean_cpu(dat);

    //Find what dim we are working in, and what depth
    int dim=-1;
    sub_dat_list sd = OPS_sub_dat_list[dat->index];
    for (int dim2 = 0; dim2 < dat->block->dims; dim2++) {
      for (int depth2 = 0; depth2 < MAX_DEPTH; depth2++) {
        if (halo == &sd->halos[MAX_DEPTH * dim2 + depth2]) {
          dim = dim2;
          break;
        }
      }
      if (dim != -1) break;
    }
    if (dim == -1) {printf("Error, hybrid could not find halo depth/dim\n"); exit(-1);}

    //if last dim, then left face on CPU, right face on GPU
    if (dim == dat->block->dims-1) {
      hybrid_offset = 0;

      //Need to determine if left or right face
      int d_m = dat->d_m[dim] + sd->d_im[dim];
      if (src_offset == (-d_m) * sd->prod[dim - 1]) { //this is the formula for the left face offset in op_mpi_rt_support.c
        ops_pack_hybrid_cpu(dat, src_offset, dest, halo->blocklength, halo->stride, halo->count);
        return;
      } // For the right face, do nothing, code below will work as usual

    } else { //otherwise need to split based on cpu_clean
      //number of halo blocks in lower dimensions
      int lowdim_count = halo->count/dat->size[dat->block->dims-1];
      hybrid_offset = lowdim_count * cpu_clean;
    }
  }

  const char *__restrict src = dat->data_d + src_offset * (OPS_soa ? dat->type_size : dat->elem_size);
  if (halo_buffer_size < halo->count * halo->blocklength * dat->dim && !OPS_gpu_direct) {
    if (halo_buffer_d != NULL)
      cutilSafeCall(hipFree(halo_buffer_d));
    cutilSafeCall(hipMalloc((void **)&halo_buffer_d,
                             halo->count * halo->blocklength * dat->dim * 4));
    halo_buffer_size = halo->count * halo->blocklength * dat->dim * 4;
  }
  char *device_buf = NULL;
  if (OPS_gpu_direct)
    device_buf = dest;
  else
    device_buf = halo_buffer_d;

  if (OPS_soa) {
    int num_threads = 128;
    int num_blocks = ((halo->blocklength * (halo->count-hybrid_offset)) - 1) / num_threads + 1;
    ops_cuda_packer_1_soa<<<num_blocks, num_threads>>>(
        src, device_buf, halo->count, halo->blocklength, halo->stride, hybrid_offset * halo->blocklength,
        dat->dim, dat->size[0]*dat->size[1]*dat->size[2]*dat->type_size);

  } else if (halo->blocklength % 4 == 0) {
    int num_threads = 128;
    int num_blocks =
        (((dat->dim * halo->blocklength / 4) * (halo->count-hybrid_offset)) - 1) / num_threads + 1;
    ops_cuda_packer_4<<<num_blocks, num_threads>>>(
        (const int *)src, (int *)device_buf, halo->count, halo->blocklength*dat->dim / 4,
        halo->stride*dat->dim / 4, hybrid_offset * (dat->dim * halo->blocklength / 4));
  } else {
    int num_threads = 128;
    int num_blocks = ((dat->dim * halo->blocklength * (halo->count-hybrid_offset)) - 1) / num_threads + 1;
    ops_cuda_packer_1<<<num_blocks, num_threads>>>(
        src, device_buf, halo->count, halo->blocklength*dat->dim,
        halo->stride*dat->dim, hybrid_offset * dat->dim * halo->blocklength);
  }

  if (!OPS_gpu_direct)
    cutilSafeCall(hipMemcpyAsync(dest + (hybrid_offset * halo->blocklength * dat->dim),
                             halo_buffer_d + (hybrid_offset * halo->blocklength * dat->dim),
                             (halo->count-hybrid_offset) * halo->blocklength * dat->dim,
                             hipMemcpyDeviceToHost, 0));

  if (ops_hybrid && hybrid_offset != 0)
    ops_pack_hybrid_cpu(dat, src_offset, dest, halo->blocklength, halo->stride, hybrid_offset);

  cutilSafeCall(hipDeviceSynchronize());
}

void ops_unpack(ops_dat dat, const int dest_offset, const char *__restrict src,
                const ops_int_halo *__restrict halo) {

  if (dat->dirty_hd == 1) {
    ops_upload_dat(dat);
    dat->dirty_hd = 0;
  }
  char *__restrict dest = dat->data_d + dest_offset * (OPS_soa ? dat->type_size : dat->elem_size);
  if (halo_buffer_size < halo->count * halo->blocklength * dat->dim && !OPS_gpu_direct) {
    if (halo_buffer_d != NULL)
      cutilSafeCall(hipFree(halo_buffer_d));
    cutilSafeCall(hipMalloc((void **)&halo_buffer_d,
                             halo->count * halo->blocklength * dat->dim * 4));
    halo_buffer_size = halo->count * halo->blocklength * dat->dim * 4;
  }

  const char *device_buf = NULL;
  if (OPS_gpu_direct)
    device_buf = src;
  else
    device_buf = halo_buffer_d;

  if (!OPS_gpu_direct)
    cutilSafeCall(hipMemcpyAsync(halo_buffer_d, src,
                             halo->count * halo->blocklength * dat->dim,
                             hipMemcpyHostToDevice,0));
  if (!ops_hybrid) cutilSafeCall(hipDeviceSynchronize());

  if (OPS_soa) {
    int num_threads = 128;
    int num_blocks = ((halo->blocklength * halo->count) - 1) / num_threads + 1;
    ops_cuda_unpacker_1_soa<<<num_blocks, num_threads>>>(
        device_buf, dest, halo->count, halo->blocklength, halo->stride,
        dat->dim, dat->size[0]*dat->size[1]*dat->size[2]*dat->type_size);
  } else if (halo->blocklength % 4 == 0) {
    int num_threads = 128;
    int num_blocks =
        (((dat->dim * halo->blocklength / 4) * halo->count) - 1) / num_threads + 1;
    ops_cuda_unpacker_4<<<num_blocks, num_threads>>>(
        (const int *)device_buf, (int *)dest, halo->count,
        halo->blocklength*dat->dim / 4, halo->stride*dat->dim / 4);
  } else {
    int num_threads = 128;
    int num_blocks = ((dat->dim * halo->blocklength * halo->count) - 1) / num_threads + 1;
    ops_cuda_unpacker_1<<<num_blocks, num_threads>>>(
        device_buf, dest, halo->count, halo->blocklength*dat->dim, halo->stride*dat->dim);
  }

  dat->dirty_hd = 2;

  if (ops_hybrid) {
    ops_unpack_hybrid_cpu(dat, dest_offset, src, halo->blocklength, halo->stride, halo->count);
    cutilSafeCall(hipDeviceSynchronize());
  }
}

char* ops_realloc_fast(char *ptr, size_t olds, size_t news) {
  if (OPS_gpu_direct) {
    if (ptr == NULL) {
      cutilSafeCall(hipMalloc((void **)&ptr, news));
      return ptr;
    } else {
      if (OPS_diags>3) printf("Warning: cuda cache realloc\n");
      char *ptr2;
      cutilSafeCall(hipMalloc((void **)&ptr2, news));
      cutilSafeCall(hipMemcpy(ptr2, ptr, olds, hipMemcpyDeviceToDevice));
      cutilSafeCall(hipFree(ptr));
      return ptr2;
    }
  } else {
    char *ptr2;
    cutilSafeCall(hipHostMalloc((void**)&ptr2,news));
    if (olds > 0)
  	  memcpy(ptr2, ptr, olds);
    if (ptr != NULL) cutilSafeCall(hipHostFree(ptr));
    return ptr2;
  }
}

__global__ void copy_kernel_tobuf(char *dest, char *src, int rx_s, int rx_e,
                                  int ry_s, int ry_e, int rz_s, int rz_e,
                                  int x_step, int y_step, int z_step,
                                  int size_x, int size_y, int size_z,
                                  int buf_strides_x, int buf_strides_y,
                                  int buf_strides_z, int type_size, int dim, int OPS_soa) {

  int idx_z = rz_s + z_step * (blockDim.z * blockIdx.z + threadIdx.z);
  int idx_y = ry_s + y_step * (blockDim.y * blockIdx.y + threadIdx.y);
  int idx_x = rx_s + x_step * (blockDim.x * blockIdx.x + threadIdx.x);

  if ((x_step == 1 ? idx_x < rx_e : idx_x > rx_e) &&
      (y_step == 1 ? idx_y < ry_e : idx_y > ry_e) &&
      (z_step == 1 ? idx_z < rz_e : idx_z > rz_e)) {

    if (OPS_soa) src += (idx_z * size_x * size_y + idx_y * size_x + idx_x) * type_size;
    else src += (idx_z * size_x * size_y + idx_y * size_x + idx_x) * type_size * dim;
    dest += ((idx_z - rz_s) * z_step * buf_strides_z +
             (idx_y - ry_s) * y_step * buf_strides_y +
             (idx_x - rx_s) * x_step * buf_strides_x) *
            type_size * dim ;
    for (int d = 0; d < dim; d++) {
      memcpy(dest+d*type_size, src, type_size);
      if (OPS_soa) src += size_x * size_y * size_z * type_size;
      else src += type_size;
    }
  }
}

__global__ void copy_kernel_frombuf(char *dest, char *src, int rx_s, int rx_e,
                                    int ry_s, int ry_e, int rz_s, int rz_e,
                                    int x_step, int y_step, int z_step,
                                    int size_x, int size_y, int size_z,
                                    int buf_strides_x, int buf_strides_y,
                                    int buf_strides_z, int type_size, int dim, int OPS_soa) {

  int idx_z = rz_s + z_step * (blockDim.z * blockIdx.z + threadIdx.z);
  int idx_y = ry_s + y_step * (blockDim.y * blockIdx.y + threadIdx.y);
  int idx_x = rx_s + x_step * (blockDim.x * blockIdx.x + threadIdx.x);

  if ((x_step == 1 ? idx_x < rx_e : idx_x > rx_e) &&
      (y_step == 1 ? idx_y < ry_e : idx_y > ry_e) &&
      (z_step == 1 ? idx_z < rz_e : idx_z > rz_e)) {

    if (OPS_soa) dest += (idx_z * size_x * size_y + idx_y * size_x + idx_x) * type_size;
    else dest += (idx_z * size_x * size_y + idx_y * size_x + idx_x) * type_size * dim;
    src += ((idx_z - rz_s) * z_step * buf_strides_z +
            (idx_y - ry_s) * y_step * buf_strides_y +
            (idx_x - rx_s) * x_step * buf_strides_x) *
           type_size * dim;
    for (int d = 0; d < dim; d++) {
      memcpy(dest, src + d * type_size, type_size);
      if (OPS_soa) dest += size_x * size_y * size_z * type_size;
      else dest += type_size;
    }
  }
}

void ops_halo_copy_tobuf(char *dest, int dest_offset, ops_dat src, int rx_s,
                         int rx_e, int ry_s, int ry_e, int rz_s, int rz_e,
                         int x_step, int y_step, int z_step, int buf_strides_x,
                         int buf_strides_y, int buf_strides_z) {

  dest += dest_offset;
  int thr_x = abs(rx_s - rx_e);
  int blk_x = 1;
  if (abs(rx_s - rx_e) > 8) {
    blk_x = (thr_x - 1) / 8 + 1;
    thr_x = 8;
  }
  int thr_y = abs(ry_s - ry_e);
  int blk_y = 1;
  if (abs(ry_s - ry_e) > 8) {
    blk_y = (thr_y - 1) / 8 + 1;
    thr_y = 8;
  }
  int thr_z = abs(rz_s - rz_e);
  int blk_z = 1;
  if (abs(rz_s - rz_e) > 8) {
    blk_z = (thr_z - 1) / 8 + 1;
    thr_z = 8;
  }

  int size =
      abs(src->elem_size * (rx_e - rx_s) * (ry_e - ry_s) * (rz_e - rz_s));
  char *gpu_ptr;
  if (OPS_gpu_direct)
    gpu_ptr = dest;
  else {
    if (halo_buffer_size < size) {
      if (halo_buffer_d != NULL)
        cutilSafeCall(hipFree(halo_buffer_d));
      cutilSafeCall(hipMalloc((void **)&halo_buffer_d, size * sizeof(char)));
      halo_buffer_size = size;
    }
    gpu_ptr = halo_buffer_d;
  }

  if (src->dirty_hd == 1) {
    ops_upload_dat(src);
    src->dirty_hd = 0;
  }

  dim3 grid(blk_x, blk_y, blk_z);
  dim3 tblock(thr_x, thr_y, thr_z);
  copy_kernel_tobuf<<<grid, tblock>>>(
      gpu_ptr, src->data_d, rx_s, rx_e, ry_s, ry_e, rz_s, rz_e, x_step, y_step,
      z_step, src->size[0], src->size[1], src->size[2], buf_strides_x,
      buf_strides_y, buf_strides_z, src->type_size, src->dim, OPS_soa);

  if (!OPS_gpu_direct)
    cutilSafeCall(hipMemcpy(dest, halo_buffer_d, size * sizeof(char),
                             hipMemcpyDeviceToHost));
}

void ops_halo_copy_frombuf(ops_dat dest, char *src, int src_offset, int rx_s,
                           int rx_e, int ry_s, int ry_e, int rz_s, int rz_e,
                           int x_step, int y_step, int z_step,
                           int buf_strides_x, int buf_strides_y,
                           int buf_strides_z) {

  src += src_offset;
  int thr_x = abs(rx_s - rx_e);
  int blk_x = 1;
  if (abs(rx_s - rx_e) > 8) {
    blk_x = (thr_x - 1) / 8 + 1;
    thr_x = 8;
  }
  int thr_y = abs(ry_s - ry_e);
  int blk_y = 1;
  if (abs(ry_s - ry_e) > 8) {
    blk_y = (thr_y - 1) / 8 + 1;
    thr_y = 8;
  }
  int thr_z = abs(rz_s - rz_e);
  int blk_z = 1;
  if (abs(rz_s - rz_e) > 8) {
    blk_z = (thr_z - 1) / 8 + 1;
    thr_z = 8;
  }

  int size =
      abs(dest->elem_size * (rx_e - rx_s) * (ry_e - ry_s) * (rz_e - rz_s));
  char *gpu_ptr;
  if (OPS_gpu_direct)
    gpu_ptr = src;
  else {
    if (halo_buffer_size < size) {
      if (halo_buffer_d != NULL)
        cutilSafeCall(hipFree(halo_buffer_d));
      cutilSafeCall(hipMalloc((void **)&halo_buffer_d, size * sizeof(char)));
      halo_buffer_size = size;
    }
    gpu_ptr = halo_buffer_d;
    cutilSafeCall(hipMemcpy(halo_buffer_d, src, size * sizeof(char),
                             hipMemcpyHostToDevice));
  }

  if (dest->dirty_hd == 1) {
    ops_upload_dat(dest);
    dest->dirty_hd = 0;
  }

  dim3 grid(blk_x, blk_y, blk_z);
  dim3 tblock(thr_x, thr_y, thr_z);
  copy_kernel_frombuf<<<grid, tblock>>>(
      dest->data_d, gpu_ptr, rx_s, rx_e, ry_s, ry_e, rz_s, rz_e, x_step, y_step,
      z_step, dest->size[0], dest->size[1], dest->size[2], buf_strides_x,
      buf_strides_y, buf_strides_z, dest->type_size, dest->dim, OPS_soa);
  dest->dirty_hd = 2;
}

#ifdef __cplusplus
}
#endif
